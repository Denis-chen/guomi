#include "hip/hip_runtime.h"
#include "GPU_SM4.h"

template<typename T>
__device__ __forceinline__ u1 get_byte(size_t byte_num, T input)
{
   return static_cast<u1>(
      input >> ((sizeof(T)-1-(byte_num&(sizeof(T)-1))) << 3)
      );
}


template<typename T>
__device__ __forceinline__ T load_be(const uint8_t in[], size_t off)
{
   in += off * sizeof(T);
   T out = 0;
   for(size_t i = 0; i != sizeof(T); ++i)
      out = static_cast<T>((out << 8) | in[i]);
   return out;
}


__device__ __forceinline__ void store_be(uint32_t in, uint8_t out[4])
{
   out[0] = get_byte(0, in);
   out[1] = get_byte(1, in);
   out[2] = get_byte(2, in);
   out[3] = get_byte(3, in);
}


template<typename T>
__device__ __forceinline__ void store_be(uint8_t out[], T x0, T x1)
{
   store_be(x0, out + (0 * sizeof(T)));
   store_be(x1, out + (1 * sizeof(T)));
}

template<typename T>
__device__ __forceinline__ void store_be(uint8_t out[], T x0, T x1, T x2, T x3)
{
   store_be(x0, out + (0 * sizeof(T)));
   store_be(x1, out + (1 * sizeof(T)));
   store_be(x2, out + (2 * sizeof(T)));
   store_be(x3, out + (3 * sizeof(T)));
}

template<size_t ROT, typename T>
__device__ __forceinline__ T rotl(T input)
{
	// static_assert(ROT > 0 && ROT < 8*sizeof(T), "Invalid rotation constant");
	return static_cast<T>((input << ROT) | (input >> (8*sizeof(T) - ROT)));
}



__device__ __forceinline__ uint32_t make_uint32(uint8_t i0, uint8_t i1, uint8_t i2, uint8_t i3)
{
   return ((static_cast<uint32_t>(i0) << 24) |
           (static_cast<uint32_t>(i1) << 16) |
           (static_cast<uint32_t>(i2) <<  8) |
           (static_cast<uint32_t>(i3)));
}


// Variant of T for key schedule
__device__ __forceinline__ u4 SM4_Tp(u4 b)
{
	const u4 t = make_uint32(Sbox[get_byte(0,b)], Sbox[get_byte(1,b)], Sbox[get_byte(2,b)], Sbox[get_byte(3,b)]);

	// L' linear transform
	return t ^ rotl<13>(t) ^ rotl<23>(t);
}


__device__ __forceinline__ u4 SM4_T(u4 b)
{
	return Sbox_T[get_byte(0,b)] ^ (Sbox_T8[get_byte(1,b)]) ^ (Sbox_T16[get_byte(2,b)]) ^ (Sbox_T24[get_byte(3,b)]);
	// return Sbox_T[get_byte(0,b)] ^ rotr<8>(Sbox_T[get_byte(1,b)]) ^ rotr<16>(Sbox_T[get_byte(2,b)]) ^ rotr<24>(Sbox_T[get_byte(3,b)]);
}


//key explansion

__global__ void SM4_key_schedule(const u1 key[SM4_KEY_SIZE], u4 rkey[SM4_RND_KEY_SIZE / sizeof(u4)])
{
	// System parameter or family key
	const u4 FK[4] = { 0xa3b1bac6, 0x56aa3350, 0x677d9197, 0xb27022dc };

	const u4 CK[32] = {
	0x00070E15, 0x1C232A31, 0x383F464D, 0x545B6269,
	0x70777E85, 0x8C939AA1, 0xA8AFB6BD, 0xC4CBD2D9,
	0xE0E7EEF5, 0xFC030A11, 0x181F262D, 0x343B4249,
	0x50575E65, 0x6C737A81, 0x888F969D, 0xA4ABB2B9,
	0xC0C7CED5, 0xDCE3EAF1, 0xF8FF060D, 0x141B2229,
	0x30373E45, 0x4C535A61, 0x686F767D, 0x848B9299,
	0xA0A7AEB5, 0xBCC3CAD1, 0xD8DFE6ED, 0xF4FB0209,
	0x10171E25, 0x2C333A41, 0x484F565D, 0x646B7279
  };

	u4 K[4];

	K[0] = load_be<u4>(key, 0) ^ FK[0];
	K[1] = load_be<u4>(key, 1) ^ FK[1];
	K[2] = load_be<u4>(key, 2) ^ FK[2];
	K[3] = load_be<u4>(key, 3) ^ FK[3];

	forloop (i, 0, 32)
	{
		K[i % 4] ^= SM4_Tp(K[(i+1)%4] ^ K[(i+2)%4] ^ K[(i+3)%4] ^ CK[i]);
		rkey[i] = K[i % 4];
	}
}

__device__ void SM4_enc_block(const u1 in[SM4_BLOCK_SIZE], u1 out[SM4_BLOCK_SIZE], const u4 rkey[SM4_RND_KEY_SIZE / sizeof(u4)])
{
  u4 B0 = load_be<u4>(in, 0);
  u4 B1 = load_be<u4>(in, 1);
  u4 B2 = load_be<u4>(in, 2);
  u4 B3 = load_be<u4>(in, 3);

  // SM4_RNDS( 0,  1,  2,  3, SM4_T_slow);
  SM4_RNDS( 0,  1,  2,  3, SM4_T);
  SM4_RNDS( 4,  5,  6,  7, SM4_T);
  SM4_RNDS( 8,  9, 10, 11, SM4_T);
  SM4_RNDS(12, 13, 14, 15, SM4_T);
  SM4_RNDS(16, 17, 18, 19, SM4_T);
  SM4_RNDS(20, 21, 22, 23, SM4_T);
  SM4_RNDS(24, 25, 26, 27, SM4_T);
  SM4_RNDS(28, 29, 30, 31, SM4_T);
  // SM4_RNDS(28, 29, 30, 31, SM4_T_slow);

  store_be(out, B3, B2, B1, B0);
}

__device__ void SM4_dec_block(const u1 in[SM4_BLOCK_SIZE], u1 out[SM4_BLOCK_SIZE], const u4 rkey[SM4_RND_KEY_SIZE / sizeof (u4)])
{
	u4 B0 = load_be<u4>(in, 0);
	u4 B1 = load_be<u4>(in, 1);
	u4 B2 = load_be<u4>(in, 2);
	u4 B3 = load_be<u4>(in, 3);

	// SM4_RNDS(31, 30, 29, 28, SM4_T_slow);
	SM4_RNDS(31, 30, 29, 28, SM4_T);
	SM4_RNDS(27, 26, 25, 24, SM4_T);
	SM4_RNDS(23, 22, 21, 20, SM4_T);
	SM4_RNDS(19, 18, 17, 16, SM4_T);
	SM4_RNDS(15, 14, 13, 12, SM4_T);
	SM4_RNDS(11, 10,  9,  8, SM4_T);
	SM4_RNDS( 7,  6,  5,  4, SM4_T);
	SM4_RNDS( 3,  2,  1,  0, SM4_T);
	// SM4_RNDS( 3,  2,  1,  0, SM4_T_slow);

	store_be(out, B3, B2, B1, B0);
}

__global__ void SM4_enc_blocks(int n_block, const u1 *in, u1 *out, const u4 *rkey)
{
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  int stride = blockDim.x*gridDim.x;

  for (int i=index; i<n_block; i +=stride)
    SM4_enc_block(in+16*i, out+16*i,rkey);
}


__global__ void SM4_dec_blocks(int n_block, const u1 *in, u1 *out, const u4 *rkey)
{
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  int stride = blockDim.x*gridDim.x;

  for (int i=index; i<n_block; i +=stride)
    SM4_dec_block(in+16*i, out+16*i, rkey);
}

void gpu_sm4_encrypt(const uint8_t *plain, const uint8_t *key, uint8_t *cipher, uint n_block)
{
  u4 *rkey;
  hipMallocManaged(&rkey,sizeof(u4)*(SM4_RND_KEY_SIZE / sizeof(u4)));

  SM4_key_schedule<<<1,1>>>(key, rkey);
	hipDeviceSynchronize();

  int blockSize = 128;
  int numBlocks = (n_block + blockSize -1) /blockSize;

  SM4_enc_blocks<<<numBlocks,blockSize>>>(n_block, plain, cipher, rkey);
  hipDeviceSynchronize();
}

void gpu_sm4_decrypt(uint8_t *plain, const uint8_t *key, const uint8_t *cipher, uint n_block)
{
  u4 *rkey;
  hipMallocManaged(&rkey,sizeof(u4)*(SM4_RND_KEY_SIZE / sizeof(u4)));

  SM4_key_schedule<<<1,1>>>(key, rkey);
	hipDeviceSynchronize();

  int blockSize = 128;
  int numBlocks = (n_block + blockSize -1) /blockSize;

  SM4_dec_blocks<<<numBlocks,blockSize>>>(n_block, cipher, plain, rkey);
  hipDeviceSynchronize();
}

void outputChar(u1 *text, size_t len)
{
    forloop( i, 0, len )
    {
        printf("0x%02x, ", text[i]);
        if (i % 16 == 15)
            puts("");
    }
    puts("");
}

void benchmark_sm4_encrypt(const uint8_t *p, const uint8_t *key, uint8_t *c, uint n_block)
{
  int turns = 100;
	clock_t t=clock();
  for(int i=0; i<turns; i++)
  {
    gpu_sm4_encrypt(p,key,c,n_block);
  }
	double tt = (double)(clock() - t)/(CLOCKS_PER_SEC*turns);
	double speed =(double) (16*n_block)/(1024*1024*tt);
	printf("SM4_encrypt>>> blocks: %d, time: %f s, speed: %f MB/s\n",n_block,tt,speed);
}

void benchmark_sm4_decrypt(uint8_t *p, const uint8_t *key, const uint8_t *c, uint n_block)
{
  int turns = 100;
	clock_t t=clock();
  for(int i=0; i<turns; i++)
  {
    gpu_sm4_decrypt(p,key,c,n_block);
  }
	double tt = (double)(clock() - t)/(CLOCKS_PER_SEC*turns);
	double speed =(double) (16*n_block)/(1024*1024*tt);
	printf("SM4_decrypt>>> blocks: %d, time: %f s, speed: %f MB/s\n",n_block,tt,speed);
}
