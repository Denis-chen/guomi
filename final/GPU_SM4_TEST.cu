#include "GPU_SM4.h"

int main()
{
	u1 * key;
	hipMallocManaged(&key,sizeof(u1)*16);
	u1 * p;
	hipMallocManaged(&p,sizeof(u1)*16*test_blocks);

	u1 *c;
	hipMallocManaged(&c,sizeof(u1)*16*test_blocks);

	u1 p2[SM4_BLOCK_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};

	u1 key2[SM4_KEY_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};
	for (int i=0; i<16; i++)
	{
		key[i] = key2[i];
	}

  for (int i=0; i<test_blocks*16; i++)
  {
    p[i] = p2[i%16];
  }

  gpu_sm4_encrypt(p,key,c,test_blocks);

  for (int k=0; k<16*2; k++)
	{
		printf("%02x ",c[k]);
	}

  for (int i=0; i<16*2; i++)
	{
		p[i] = 0x01;
	}

  printf("\n");

  gpu_sm4_decrypt(p,key,c,test_blocks);

  for (int k=0; k<16*2; k++)
	{
		printf("%02x ",p[k]);
	}

}
