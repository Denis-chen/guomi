#include "GPU_SM4.h"

#define test_blocks (8)

int main()
{
	u1 * key;
	hipMallocManaged(&key,sizeof(u1)*16);
	u1 * p;
	hipMallocManaged(&p,sizeof(u1)*16*test_blocks);

	u1 *c;
	hipMallocManaged(&c,sizeof(u1)*16*test_blocks);

	u1 p2[SM4_BLOCK_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};

	u1 key2[SM4_KEY_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};
	for (int i=0; i<16; i++)
	{
		key[i] = key2[i];
	}

  	for (int i=0; i<test_blocks*16; i++)
  	{
    	p[i] = p2[i%16];
  	}

  	gpu_sm4_encrypt(p,key,c,test_blocks);

  	outputChar(c, 16);

  	gpu_sm4_decrypt(p,key,c,test_blocks);

  	outputChar(p, 16);

}
