#include "hip/hip_runtime.h"
#include "sm4_share.h"

int main(int argc, char * argv[])
{
  InitCUDA();

  char * filename = argv[1];
  char * outfilename = argv[2];
  int fd = open(filename,O_RDONLY);
  int fdw = open(outfilename,O_WRONLY|O_CREAT,S_IRUSR|S_IWUSR);
  if (!fd)
  {
    printf("%s\n", "open file error");
    exit(-1);
  }

  u1 * key;
	hipMallocManaged(&key,sizeof(u1)*16);
	u4 * rkey;
	hipMallocManaged(&rkey,sizeof(u4)*(SM4_RND_KEY_SIZE / sizeof(u4)));
  u1 * p;
  hipMallocManaged(&p,sizeof(u1)*SIZE);

  u1 *c;
  hipMallocManaged(&c,sizeof(u1)*SIZE);

  u1 key2[SM4_KEY_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};
	for (int i=0; i<16; i++)
	{
		key[i] = key2[i];
	}

  SM4_key_schedule<<<1,1>>>( key, rkey );
	hipDeviceSynchronize();

  int haveread;


  haveread=read(fd,p,SIZE);
  printf("%d\n",haveread);
  clock_t t=clock();
  int blockSize = 512;
  int blocks = haveread/16;
  int numBlocks = (blocks + blockSize -1) /blockSize;
  speed_test_share<<<numBlocks,blockSize>>>(blocks,p,c,rkey);
  hipDeviceSynchronize();
  double tt = (double)(clock() - t)/CLOCKS_PER_SEC;
  double speed =(double) (haveread)/(1024*1024*tt);
  write(fdw,c,haveread);
  close(fdw);



	printf("time: %f s\nspeed: %f \n",tt,speed);

	// printf("hahahaha\n");

	// for (int j=0 ; j<32; j++)
	// {
	// 	printf("%04x,",rkey[j]);
	// }



}
