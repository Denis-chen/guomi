#include "hip/hip_runtime.h"
#include "sm4_share.h"


void test(u1 *p, u1*c, u4 *rkey, int blockSize)
{
  clock_t t=clock();

	// speed_test_share<<<1,1>>>(1,p, c, rkey );
  // int blockSize = 512;
  int numBlocks = (test_blocks + blockSize -1) /blockSize;
  speed_test_share<<<numBlocks,blockSize>>>(test_blocks,p,c,rkey);

	hipDeviceSynchronize();
  double tt = (double)(clock() - t)/CLOCKS_PER_SEC;
  double speed =(double) (16*test_blocks)/(1024*1024*tt);

	printf("thrads: %d \ntime: %f s\nspeed: %f \n",blockSize,tt,speed);
}

int main()
{
  InitCUDA();
	u1 * key;
	hipMallocManaged(&key,sizeof(u1)*16);
	u4 * rkey;
	hipMallocManaged(&rkey,sizeof(u4)*(SM4_RND_KEY_SIZE / sizeof(u4)));
	u1 * p;
	hipMallocManaged(&p,sizeof(u1)*16*test_blocks);

	u1 *c;
	hipMallocManaged(&c,sizeof(u1)*16*test_blocks);

	u1 p2[SM4_BLOCK_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};

	u1 key2[SM4_KEY_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};
	for (int i=0; i<16; i++)
	{
		key[i] = key2[i];
	}

  for (int i=0; i<test_blocks*16; i++)
  {
    p[i] = p2[i%16];
  }



	SM4_key_schedule<<<1,1>>>( key, rkey );
	hipDeviceSynchronize();

  test(p,c,rkey,32);
  test(p,c,rkey,64);
  test(p,c,rkey,128);
  test(p,c,rkey,256);
  test(p,c,rkey,512);
  test(p,c,rkey,1024);
  // test(p,c,rkey,test_blocks);
	// printf("hahahaha\n");

	// for (int j=0 ; j<32; j++)
	// {
	// 	printf("%04x,",rkey[j]);
	// }



	//for (int k=0; k<16*test_blocks; k++)
	//{
	//	printf("%02x ",c[k]);
	//}
}
